
#include <hip/hip_runtime.h>
#define FLOAT_TO_BITS(x) (*reinterpret_cast<unsigned int*>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float*>(x))

__device__ __forceinline__ unsigned int extract_exponent(float *a) {
  unsigned int temp = *(reinterpret_cast<unsigned int*>(a));
  temp = (temp << 1 >> 24); // single preciision, 1 sign bit, 23 mantissa bits
  return temp-127+1; // exponent offset and virtual bit
}

__device__ __forceinline__ unsigned int round_bitwise_stochastic(unsigned int target,
                                                                 unsigned int rand_prob,
                                                                 int man_bits) {
    unsigned int mask = (1 << (23-man_bits)) - 1;
    unsigned int add_r = target+(rand_prob & mask);
    unsigned int quantized = add_r & ~mask;
    return quantized;
}

__device__ __forceinline__ unsigned int round_bitwise_nearest(unsigned int target,
                                                              int man_bits) {
    unsigned int mask = (1 << (23-man_bits)) - 1;
    unsigned int rand_prob = 1 << (23-man_bits-1);
    unsigned int add_r = target+rand_prob;
    unsigned int quantized = add_r & ~mask;
    return quantized;
}

__device__ __forceinline__ unsigned int clip_exponent(int exp_bits, int man_bits,
                                                      unsigned int old_num,
                                                      unsigned int quantized_num) {
  int quantized_exponent_store = quantized_num << 1 >> 1 >> 23; // 1 sign bit, 23 mantissa bits
  int min_exponent_store = -((1 << (exp_bits-1))-1) + 126;
  int max_exponent_store = 1 << (exp_bits-1) + 126;
  if (quantized_exponent_store > max_exponent_store) {
    unsigned int max_man = (unsigned int ) -1 << 9 >> 9 >> (23-man_bits) << (23-man_bits); // 1 sign bit, 8 exponent bits, 1 virtual bit
    unsigned int max_num = ((unsigned int) max_exponent_store << 23) | max_man;
    unsigned int old_sign = old_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  } else if (quantized_exponent_store < min_exponent_store) {
    unsigned int min_num = ((unsigned int) min_exponent_store << 23);
    unsigned int old_sign = old_num >> 31 << 31;
    quantized_num = old_sign | min_num;
  }
  return quantized_num;
}